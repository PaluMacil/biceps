// Dan Wolf


#include <hip/hip_runtime.h>
#include <iostream>
#include <string>
#include <chrono>

// https://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api/14038590#14038590
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void multMat(int n, int *arrForce_d, int *arrDistance_d, int *arrAnswer_d) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n){
        arrAnswer_d[i] = arrForce_d[i] * arrDistance_d[i];
    }
}

int main(int argc, char **argv) {
    auto n = atoi(argv[1]);
    size_t bytes = n* sizeof(int);

    // host pointers
    int* arrForce;
    int* arrDistance;
    int* arrAnswer;

    // device pointers
    int* arrForce_d;
    int* arrDistance_d;
    int* arrAnswer_d;

    // allocate on host
    arrForce = (int*)malloc(bytes);
    arrDistance = (int*)malloc(bytes);
    arrAnswer = (int*)malloc(bytes);
    // initialize on host, at n=16, this initializes to
    // (1, 2, 3, 4, 5, 6, 7, 8, 9, 8, 7, 6, 5, 4, 3, 2)
    // (1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 1, 2, 3, 4, 5)
    // Answer: 1 + 4 + 9 + 16 + 25 + 36 + 49 + 64 + 81 + 80 + 77 + 6 + 10 + 12 + 12 + 10 = 492
    int forceValue = 1;
    int distanceValue = 0;
    for (int i = 0; i < n; i++){
        arrForce[i] = forceValue;
        if (i < n/2) {
            forceValue = forceValue + 1;
        } else {
            forceValue = forceValue - 1;
        }

        distanceValue = distanceValue > 10 ? distanceValue - 10 : distanceValue + 1;
        arrDistance[i] = distanceValue;
    }

    // allocate on device

    // print
    std::cout << "force: ";
    for (int i = 0; i < n; i++){
        std::cout << arrForce[i] << " ";
    }
    std::cout << '\n' << "dist: ";
    for (int i = 0; i < n; i++){
        std::cout << arrDistance[i] << " ";
    }

    std::chrono::high_resolution_clock::time_point t1 = std::chrono::high_resolution_clock::now();
    // if more than the number of elements is passed in, don't use the gpu
    if (argc > 2) {
        std::cout << '\n' << "using CPU" << '\n';
        for (int i = 0; i < n; i++) {
            arrAnswer[i] = arrForce[i] * arrDistance[i];
        }
    } else {
        std::cout << '\n' << "using GPU" << '\n';
        const int BLOCK_SIZE = 1024;
        dim3 dimBlock (BLOCK_SIZE);
        dim3 dimGrid = (int)ceil((float)n / BLOCK_SIZE);

        gpuErrchk(hipMalloc(&arrForce_d, bytes));
        gpuErrchk(hipMalloc(&arrDistance_d, bytes));
        gpuErrchk(hipMalloc(&arrAnswer_d, bytes));

        gpuErrchk(hipMemcpy(arrForce_d, arrForce, bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(arrDistance_d, arrDistance, bytes, hipMemcpyHostToDevice));
        gpuErrchk(hipMemcpy(arrAnswer_d, arrAnswer, bytes, hipMemcpyHostToDevice));

        multMat<<<dimGrid, dimBlock>>>(n, arrForce_d, arrDistance_d, arrAnswer_d);
        gpuErrchk(hipPeekAtLastError());

        gpuErrchk(hipMemcpy(arrAnswer, arrAnswer_d, bytes, hipMemcpyDeviceToHost));

        gpuErrchk(hipFree(arrForce_d))
        gpuErrchk(hipFree(arrDistance_d))
        gpuErrchk(hipFree(arrAnswer_d))
    }
    std::chrono::high_resolution_clock::time_point t2 = std::chrono::high_resolution_clock::now();

    int total = 0;
    for (int i = 0; i < n; i++) {
        total = total + arrAnswer[i];
    }
    std::cout << "answer: " << total << '\n' << '\n';
    std::cout << "time for calculation: " << (t2 - t1).count() << "ns";

    return 0;
}
